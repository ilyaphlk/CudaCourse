#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>

__global__ void hello() {
    printf("hello, world!\n");
    fflush(stdout);
}

int main() {
    hello<<<1, 1>>>();

    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    return 0;
}

