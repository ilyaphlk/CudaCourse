#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>

__global__ void hello(char* buffer) {
    sprintf(buffer, "hello, world! thread number %d\n", threadIdx.x);
}

int main() {
    char* d_buffer;
    hipMalloc(&d_buffer, 256);

    hello<<<1, 1>>>(d_buffer);

    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    char h_buffer[256];
    hipMemcpy(h_buffer, d_buffer, 256, hipMemcpyDeviceToHost);
    printf("%s", h_buffer);

    hipFree(d_buffer);

    return 0;
}

