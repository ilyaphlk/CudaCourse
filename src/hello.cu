#include <cstdio>
#include <hip/hip_runtime.h>

__global__ void hello() {
    printf("hello, world!\n", threadIdx.x);
}

int main() {
    hello<<<1, 1>>>();

    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    return 0;
}

