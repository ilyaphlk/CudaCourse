#include <stdio.h>
#include <hip/hip_runtime.h>

__device__ void write_string(char* buffer, const char* message) {
    int i = 0;
    while (message[i] != '\0') {
        buffer[i] = message[i];
        ++i;
    }
    buffer[i] = '\0';
}

__global__ void hello(char* buffer) {
    write_string(buffer, "hello, world!\n");
}

int main() {
    char* d_buffer;
    hipMalloc(&d_buffer, 256);

    hello<<<1, 1>>>(d_buffer);

    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    char h_buffer[256];
    hipMemcpy(h_buffer, d_buffer, 256, hipMemcpyDeviceToHost);
    printf("%s", h_buffer);

    FILE *f = fopen("output.txt", "w");
    fprintf(f, "%s\n", h_buffer);
    fclose(f);

    hipFree(d_buffer);

    return 0;
}

