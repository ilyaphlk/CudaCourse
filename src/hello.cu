#include <cstdio>
#include <hip/hip_runtime.h>

__global__ void hello() {
    printf("hello, world! thread number %d\n", threadIdx.x);
}

int main() {
    hello<<<1, 1>>>();

    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    return 0;
}

