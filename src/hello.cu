
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cuda_device_runtime_api.h>

__global__ void hello() {
    printf("hello, world!\n");
}

int main() {
    hello<<<1, 1>>>();

    hipDeviceSynchronize();

    return 0;
}

