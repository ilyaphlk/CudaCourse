#include <cstdio>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>

__global__ void hello() {
    printf("hello, world!\n");
}

int main() {
    hello<<<1, 1>>>();

    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    return 0;
}

