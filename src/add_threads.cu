#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void add_threads(int* s, const int n) {
    *s = threadIdx.x + n * blockIdx.x;
}

int main() {
    int hSum = 0;
    int* dSum;
    CudaMalloc(&dSum, sizeof(int));
    const int n = 8;
    add_threads<<<1, n>>>(&dSum, n);

    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    CudaMemcpy(&hSum, dSum);
    CudaFree(dSum);

    printf("sum is %d\n", hSum);
}
