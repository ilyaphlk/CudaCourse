#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void add_threads(int* s, const int n) {
    //*s = threadIdx.x + n * blockIdx.x;
    *s = n;
}

int main() {
    int sum = 0;
    const int n = 8;
    add_threads<<<n, n>>>(&sum, n);

    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    printf("sum is %d\n", sum);
}
