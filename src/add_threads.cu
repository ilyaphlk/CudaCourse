#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void add_threads(int* s) {
    *s = threadIdx.x + blockIdx.x;
}

int main() {
    int sum = 0;
    add_threads<<<2, 2>>>(&sum);

    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    printf("sum is %d\n", sum);
}
