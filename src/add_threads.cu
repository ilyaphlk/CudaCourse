#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void add_threads(int* s, const int n) {
    *s = threadIdx.x + n * blockIdx.x;
}

int main() {
    int hSum = 0;
    int* dSum;
    hipMalloc(&dSum, sizeof(int));
    const int n = 8;
    add_threads<<<1, n>>>(&dSum, n);

    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    hipMemcpy(&hSum, dSum);
    hipFree(dSum);

    printf("sum is %d\n", hSum);
}
